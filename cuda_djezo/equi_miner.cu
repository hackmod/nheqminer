#include "hip/hip_runtime.h"
/*
  Equihash solver created by djeZo (l33tsoftw@gmail.com) for NiceHash

  Based on CUDA solver by John Tromp released under MIT license.

  Some helper functions taken out of OpenCL solver by Marc Bevand
  released under MIT license.

  cuda_djezo solver is released by NiceHash (www.nicehash.com) under
  GPL 3.0 license. If you don't have a copy, you can obtain one from
  https://www.gnu.org/licenses/gpl-3.0.txt
*/

/*
The MIT License (MIT)

Copyright (c) 2016 John Tromp

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software, and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
The MIT License (MIT)

Copyright (c) 2016 Marc Bevand

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software, and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#ifdef WIN32
#include <Windows.h>
#endif
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>
#include <mutex>

#include "eqcuda.hpp"
#include "sm_32_intrinsics.h"

#define WN	200
#define WK	9
#define NDIGITS		(WK+1)
#define DIGITBITS	(WN/(NDIGITS))
#define PROOFSIZE (1<<WK)
#define BASE (1<<DIGITBITS)
#define NHASHES (2*BASE)
#define HASHESPERBLAKE (512/WN)
#define HASHOUT (HASHESPERBLAKE*WN/8)
#define NBLOCKS ((NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE)
#define BUCKBITS (DIGITBITS - RB)
#define NBUCKETS (1 << BUCKBITS)
#define BUCKMASK (NBUCKETS - 1)
#define SLOTBITS (RB + 2)
#define SLOTRANGE (1 << SLOTBITS)
#define NSLOTS SM
#define SLOTMASK (SLOTRANGE - 1)
#define NRESTS (1 << RB)
#define RESTMASK (NRESTS - 1)
#define CANTORBITS (2 * SLOTBITS - 2)
#define CANTORMASK ((1 << CANTORBITS) - 1)
#define CANTORMAXSQRT (2 * NSLOTS)
#define RB8_NSLOTS 640
#define RB8_NSLOTS_LD 624
#define FD_THREADS 256

// reduce vstudio warnings (__byteperm, blockIdx...)
#ifdef __INTELLISENSE__
#include <hip/device_functions.h>
#include <>
#define __launch_bounds__(max_tpb, min_blocks)
#define __CUDA_ARCH__ 520
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __shfl(uint32_t x, uint32_t y, uint32_t z);
uint32_t atomicExch(uint32_t *x, uint32_t y);
uint32_t atomicAdd(uint32_t *x, uint32_t y);
void __syncthreads(void);
void __threadfence(void);
void __threadfence_block(void);
uint32_t __ldg(const uint32_t* address);
uint64_t __ldg(const uint64_t* address);
uint4 __ldca(const uint4 *ptr);
u32 __ldca(const u32 *ptr);
u32 umin(const u32, const u32);
u32 umax(const u32, const u32);
#endif

#ifdef DEBUG
#define DEBUG_PRINT(...) do {printf(__VA_ARGS__);} while(false)
#define DEBUG_PRINT_IF(x, ...) do {if (x) printf(__VA_ARGS__);} while(false)
#else
#define DEBUG_PRINT(...)
#define DEBUG_PRINT_IF(x, ...)
#endif

#define PRECALC

static __constant__ uint32_t __align__(16) d_blake_h[16];
#ifdef PRECALC
static __constant__ uint64_t __align__(16) precalcvalues[16];
#endif

typedef u32 proof[PROOFSIZE];


struct __align__(32) slot
{
	u32 hash[8];
};


struct __align__(16) slotsmall
{
	u32 hash[4];
};


struct __align__(8) slottiny
{
	u32 hash[2];
};


template <u32 RB, u32 SM>
struct equi
{
	slot round0trees[4096][RB8_NSLOTS];
	slot trees[1][NBUCKETS][NSLOTS];
	struct
	{
		slotsmall treessmall[NSLOTS];
		slottiny treestiny[NSLOTS];
	} round2trees[NBUCKETS];
	struct
	{
		slotsmall treessmall[NSLOTS];
		slottiny treestiny[NSLOTS];
	} round3trees[NBUCKETS];
	slotsmall treessmall[4][NBUCKETS][NSLOTS];
	u32 round4bidandsids[NBUCKETS][NSLOTS];
	slottiny treestiny[1][4096][RB8_NSLOTS_LD];
	struct
	{
		u32 nslots0[4096];
		u32 nslots[9][NBUCKETS];
		u32 nslots8[4096];
	} edata;
	scontainerreal srealcont;
};


#ifndef PRECALC
__device__ __constant__ const u64 blake_iv[] =
{
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};

#else
#define ROTR64(x, n)  (((x) >> (n)) | ((x) << (64 - (n))))
__host__
static void precalc(uint64_t* message)
{
	uint64_t blake_iv[] = {
		0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
		0x510e527fade682d1, 0x9b05688c2b3e6c1f,
		0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
	};

#define SWAP64(x)  (((x) << (32)) | ((x) >> (64 - (32))))
	uint64_t v[16];

	v[0] = message[0];
	v[1] = message[1];
	v[2] = message[2];
	v[3] = message[3];
	v[4] = message[4];
	v[5] = message[5];
	v[6] = message[6];
	v[7] = message[7];
	v[8] = blake_iv[0];
	v[9] = blake_iv[1];
	v[10] = blake_iv[2];
	v[11] = blake_iv[3];
	v[12] = blake_iv[4] ^ (128 + 16);
	v[13] = blake_iv[5];
	v[14] = blake_iv[6] ^ 0xffffffffffffffffu;
	v[15] = blake_iv[7];

	v[0] = v[0] + v[4];
	v[12] = SWAP64(v[12] ^ v[0]);
	v[8] = v[8] + v[12];
	v[4] = ROTR64(v[4] ^ v[8], 24);
	//v[0] = v[0] + v[4] + m64;
	//u[12] = ROR16(u[12] ^ u[0]);
	//v[8] = v[8] + v[12];
	//u[4] = ROR2(u[4] ^ u[8], 63);
	v[1] = v[1] + v[5];
	v[2] = v[2] + v[6];
	v[3] = v[3] + v[7];
	v[13] = SWAP64(v[13] ^ v[1]);
	v[14] = SWAP64(v[14] ^ v[2]);
	v[15] = SWAP64(v[15] ^ v[3]);
	v[9] = v[9] + v[13];
	v[10] = v[10] + v[14];
	v[11] = v[11] + v[15];
	v[5] = ROTR64(v[5] ^ v[9], 24);
	v[6] = ROTR64(v[6] ^ v[10], 24);
	v[7] = ROTR64(v[7] ^ v[11], 24);
	v[1] = v[1] + v[5];
	v[2] = v[2] + v[6];
	v[3] = v[3] + v[7];
	v[13] = ROTR64(v[13] ^ v[1], 16);
	v[14] = ROTR64(v[14] ^ v[2], 16);
	v[15] = ROTR64(v[15] ^ v[3], 16);
	v[9] = v[9] + v[13];
	v[10] = v[10] + v[14];
	v[11] = v[11] + v[15];
	v[5] = ROTR64(v[5] ^ v[9], 63);
	v[6] = ROTR64(v[6] ^ v[10], 63);
	v[7] = ROTR64(v[7] ^ v[11], 63);;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(precalcvalues), v, sizeof(v), 0, hipMemcpyHostToDevice));
}
#endif

__device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b)
{
	return make_uint2(a.x ^ b.x, a.y ^ b.y);
}

__device__ __forceinline__ uint4 operator^ (uint4 a, uint4 b)
{
	return make_uint4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w);
}

__device__ __forceinline__ uint2 ROR2(const uint2 a, const int offset) 
{
	uint2 result;
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
	return result;
}

__device__ __forceinline__ uint64_t MAKE_ULONGLONG(uint32_t LO, uint32_t HI)
{
#ifdef __CUDA_ARCH__
        uint64_t result;
        asm("mov.b64	%0,{%1,%2}; \n\t"
                : "=l"(result) : "r"(LO), "r"(HI));
        return result;
#else
	return (uint64_t)LO | (((uint64_t)HI) << 32);
#endif
}

static __host__ __device__ __forceinline__ uint2 vectorize(uint64_t v) {
	uint2 result;
#ifdef __CUDA_ARCH__
	asm("// vectorize\n\t");
	asm("mov.b64	{%0,%1},%2; \n\t"
		: "=r"(result.x), "=r"(result.y) : "l"(v));
#else
	result.x = (uint32_t)(v);
	result.y = (uint32_t)(v >> 32);
#endif
	return result;
}

static __host__ __device__ __forceinline__ uint64_t devectorize(uint2 v) {
#ifdef __CUDA_ARCH__
	return MAKE_ULONGLONG(v.x, v.y);
#else
	return (((uint64_t)v.y) << 32) + v.x;
#endif
}

static __device__ __forceinline__ uint2 operator+ (uint2 a, uint2 b) {
#ifdef __CUDA_ARCH__
	uint2 result;
	asm("{\n\t"
		"add.cc.u32 %0,%2,%4; \n\t"
		"addc.u32 %1,%3,%5;   \n\t"
	"}\n\t"
		: "=r"(result.x), "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
	return result;
#else
	return vectorize(devectorize(a) + devectorize(b));
#endif
}
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) { a = a + b; }

__device__ __forceinline__ uint2 SWAPUINT2(uint2 value) 
{
	return make_uint2(value.y, value.x);
}

__device__ __forceinline__ uint2 ROR24(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x2107);
	result.y = __byte_perm(a.y, a.x, 0x6543);
	return result;
}

__device__ __forceinline__ uint2 ROR16(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x1076);
	result.y = __byte_perm(a.y, a.x, 0x5432);
	return result;
}

#undef xor3
__device__ __forceinline__
uint2 xor3(uint2 a, uint2 b, uint2 c)
{
	uint2 result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm volatile ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.x) : "r"(a.x), "r"(b.x), "r"(c.x)); // 0x96 = 0xF0 ^ 0xCC ^ 0xAA
	asm volatile ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.y) : "r"(a.y), "r"(b.y), "r"(c.y)); // 0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result.x = a.x ^ b.x;
	result.y = a.y ^ b.y;
#endif
	return result;
}

__device__ __forceinline__
uint32_t xor3(uint32_t a, uint32_t b, uint32_t c)
{
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b), "r"(c)); // 0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a ^ b ^ c;
#endif
	return result;
}

#if (CUDA_VERSION >= 9000)
#define SHFL(x, lane) __shfl_sync(0xffffffff, (x), (lane))
#define ANY(predicate) __any_sync(0xffffffff, (predicate))
#else
#define SHFL(x, lane) __shfl((x), (lane))
#define ANY(predicate) __any(predicate)
#endif

#define OPT_ASM

#ifdef USE_ADD64
#define Gn1(a, b, c, d) \
	v[a] = v[a] + v[b]; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	v[a] = v[a] + v[b]; \
	u[d] = ROR16(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn1x(a, b, c, d, x) \
	v[a] = v[a] + v[b] + x ## 64; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	v[a] = v[a] + v[b]; \
	u[d] = ROR16(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn1y(a, b, c, d, y) \
	v[a] = v[a] + v[b]; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	v[a] = v[a] + v[b] + y ## 64; \
	u[d] = ROR16(u[d] ^ u[a]); \
	v[c] = v[c] + v[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn2(a1, b1, c1, d1, a2, b2, c2, d2) \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2];  \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]); \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U); \

#define Gn3(a1, b1, c1, d1, a2, b2, c2, d2, a3, b3, c3, d3) \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2];             v[a3] = v[a3] + v[b3]; \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]);  u[d3] = SWAPUINT2(u[d3] ^ u[a3]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2];             v[c3] = v[c3] + v[d3]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]);      u[b3] = ROR24(u[b3] ^ u[c3]); \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2];             v[a3] = v[a3] + v[b3]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]);      u[d3] = ROR16(u[d3] ^ u[a3]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2];             v[c3] = v[c3] + v[d3]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U);  u[b3] = ROR2(u[b3] ^ u[c3], 63U); \

#define Gn4(a1, b1, c1, d1, a2, b2, c2, d2, a3, b3, c3, d3, a4, b4, c4, d4) \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2];             v[a3] = v[a3] + v[b3];             v[a4] = v[a4] + v[b4]; \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]);  u[d3] = SWAPUINT2(u[d3] ^ u[a3]);  u[d4] = SWAPUINT2(u[d4] ^ u[a4]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2];             v[c3] = v[c3] + v[d3];             v[c4] = v[c4] + v[d4]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]);      u[b3] = ROR24(u[b3] ^ u[c3]);      u[b4] = ROR24(u[b4] ^ u[c4]); \
	v[a1] = v[a1] + v[b1];             v[a2] = v[a2] + v[b2];             v[a3] = v[a3] + v[b3];             v[a4] = v[a4] + v[b4]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]);      u[d3] = ROR16(u[d3] ^ u[a3]);      u[d4] = ROR16(u[d4] ^ u[a4]); \
	v[c1] = v[c1] + v[d1];             v[c2] = v[c2] + v[d2];             v[c3] = v[c3] + v[d3];             v[c4] = v[c4] + v[d4]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U);  u[b3] = ROR2(u[b3] ^ u[c3], 63U);  u[b4] = ROR2(u[b4] ^ u[c4], 63U); \

#else

#define Gn1(a, b, c, d) \
	u[a] = u[a] + u[b]; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	u[a] = u[a] + u[b]; \
	u[d] = ROR16(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn1x(a, b, c, d, x) \
	u[a] = u[a] + u[b] + x; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	u[a] = u[a] + u[b]; \
	u[d] = ROR16(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn1y(a, b, c, d, y) \
	u[a] = u[a] + u[b]; \
	u[d] = SWAPUINT2(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR24(u[b] ^ u[c]); \
	u[a] = u[a] + u[b] + y; \
	u[d] = ROR16(u[d] ^ u[a]); \
	u[c] = u[c] + u[d]; \
	u[b] = ROR2(u[b] ^ u[c], 63U); \

#define Gn2(a1, b1, c1, d1, a2, b2, c2, d2) \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2];  \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]); \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U); \

#define Gn3(a1, b1, c1, d1, a2, b2, c2, d2, a3, b3, c3, d3) \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2];             u[a3] = u[a3] + u[b3]; \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]);  u[d3] = SWAPUINT2(u[d3] ^ u[a3]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2];             u[c3] = u[c3] + u[d3]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]);      u[b3] = ROR24(u[b3] ^ u[c3]); \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2];             u[a3] = u[a3] + u[b3]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]);      u[d3] = ROR16(u[d3] ^ u[a3]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2];             u[c3] = u[c3] + u[d3]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U);  u[b3] = ROR2(u[b3] ^ u[c3], 63U); \

#define Gn4(a1, b1, c1, d1, a2, b2, c2, d2, a3, b3, c3, d3, a4, b4, c4, d4) \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2];             u[a3] = u[a3] + u[b3];             u[a4] = u[a4] + u[b4]; \
	u[d1] = SWAPUINT2(u[d1] ^ u[a1]);  u[d2] = SWAPUINT2(u[d2] ^ u[a2]);  u[d3] = SWAPUINT2(u[d3] ^ u[a3]);  u[d4] = SWAPUINT2(u[d4] ^ u[a4]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2];             u[c3] = u[c3] + u[d3];             u[c4] = u[c4] + u[d4]; \
	u[b1] = ROR24(u[b1] ^ u[c1]);      u[b2] = ROR24(u[b2] ^ u[c2]);      u[b3] = ROR24(u[b3] ^ u[c3]);      u[b4] = ROR24(u[b4] ^ u[c4]); \
	u[a1] = u[a1] + u[b1];             u[a2] = u[a2] + u[b2];             u[a3] = u[a3] + u[b3];             u[a4] = u[a4] + u[b4]; \
	u[d1] = ROR16(u[d1] ^ u[a1]);      u[d2] = ROR16(u[d2] ^ u[a2]);      u[d3] = ROR16(u[d3] ^ u[a3]);      u[d4] = ROR16(u[d4] ^ u[a4]); \
	u[c1] = u[c1] + u[d1];             u[c2] = u[c2] + u[d2];             u[c3] = u[c3] + u[d3];             u[c4] = u[c4] + u[d4]; \
	u[b1] = ROR2(u[b1] ^ u[c1], 63U);  u[b2] = ROR2(u[b2] ^ u[c2], 63U);  u[b3] = ROR2(u[b3] ^ u[c3], 63U);  u[b4] = ROR2(u[b4] ^ u[c4], 63U); \

#endif

struct packer_default
{
	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB, const u32 SM)
	{
#ifdef OPT_ASM
		u32 ret;
		asm volatile (
			"vshl.u32.u32.u32.clamp.add %0, %1, %2, %3;\n\t"
			"vshl.u32.u32.u32.clamp.add %0, %0, %2, %4;"
			: "=r"(ret)
			: "r"(bucketid), "r"(SLOTBITS), "r"(s0), "r"(s1));
		return ret;
#else
		return (((bucketid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
		// BUCKMASK-ed to prevent illegal memory accesses in case of memory errors
#ifdef OPT_ASM
		u32 ret;
		asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(bid), "r"(2 * SLOTBITS), "r"(BUCKBITS));
		return ret;
#else
		return (bid >> (2 * SLOTBITS)) & BUCKMASK;
#endif
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return bid & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
#ifdef OPT_ASM
		u32 s1;
		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(s1) : "r"(bid), "r"(SLOTBITS), "r"(SLOTBITS));
		return s1;
#else
		return (bid >> SLOTBITS) & SLOTMASK;
#endif
	}
};


struct packer_cantor
{
	__device__ __forceinline__ static u32 cantor(const u32 s0, const u32 s1)
	{
		u32 a = umax(s0, s1);
		u32 b = umin(s0, s1);
#ifdef OPT_ASM
		u32 c;
		asm volatile (
			"{\n\t.reg .u32 c;\n\t"
			"mad.lo.s32     c, %1, %1, %1;\n\t"
			"vshr.u32.u32.u32.clamp.add %0, c, 1, %2;\n\t}"
			: "=r"(c)
			: "r"(a), "r"(b));
		return c;
#else
		return a * (a + 1) / 2 + b;
#endif
	}

	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB, const u32 SM)
	{
#ifdef OPT_ASM
		u32 ret;
		u32 a = umax(s0, s1);
		u32 b = umin(s0, s1);
		asm volatile (
			"{\n\t.reg .u32 c, d;\n\t"
			"mad.lo.s32     c, %3, %3, %3;\n\t"
			"vshr.u32.u32.u32.clamp.add d, c, 1, %4;\n\t"
			"vshl.u32.u32.u32.clamp.add %0, %1, %2, d;\n\t}"
			: "=r"(ret)
			: "r"(bucketid), "r"(CANTORBITS), "r"(a), "r"(b));
		return ret;
#else
		return (bucketid << CANTORBITS) | cantor(s0, s1);
#endif
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
#ifdef OPT_ASM
		u32 s1;
		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(s1) : "r"(bid), "r"(CANTORBITS), "r"(BUCKBITS));
		return s1;
#else
		return (bid >> CANTORBITS) & BUCKMASK;
#endif
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return ((bid & CANTORMASK) - cantor(0, s1)) & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
		u32 k, q, sqr;
#ifdef OPT_ASM
                asm volatile (
                        "vshl.u32.u32.u32.clamp.add %0, %1, 3, 1;"
                        : "=r"(sqr)
                        : "r"(bid & CANTORMASK));
#else
		sqr = 8 * (bid & CANTORMASK) + 1;
#endif
		// this k=sqrt(sqr) computing loop averages 3.4 iterations out of maximum 9
		for (k = CANTORMAXSQRT; (q = sqr / k) < k; k = (k + q) >> 1);
		return ((k - 1) >> 1) & SLOTMASK;
	}
};


template <u32 RB, u32 SM, typename PACKER>
__global__ void digit_first(equi<RB, SM>* eq, u32 nonce)
{
	const u32 block = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ uint2 hash_h[8];
	u32* hash_h32 = (u32*)hash_h;

	if (threadIdx.x < 16)
		hash_h32[threadIdx.x] = d_blake_h[threadIdx.x];

	__syncthreads();

	union
	{
		u64 v[16];
		uint2 u[16];
	};
	const uint2 m = make_uint2(nonce, block);
	uint64_t m64 = devectorize(m);

#ifndef PRECALC
	*(uint4*)&u[0] = *(uint4*)&hash_h[0];
	*(uint4*)&u[2] = *(uint4*)&hash_h[2];
	*(uint4*)&u[4] = *(uint4*)&hash_h[4];
	*(uint4*)&u[6] = *(uint4*)&hash_h[6];
	*(uint4*)&u[8] = *(uint4*)&blake_iv[0];
	*(uint4*)&u[10] = *(uint4*)&blake_iv[2];
	*(uint4*)&u[12] = *(uint4*)&blake_iv[4];
	*(uint4*)&u[14] = *(uint4*)&blake_iv[6];
	v[12] = v[12] ^ (128 + 16);
	v[14] = v[14] ^ 0xffffffffffffffffu;

	// mix 1-a
	Gn1y(0, 4,  8, 12, m);
	Gn3(              1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
#else

	*(uint4*)&u[0] = *(uint4*)&precalcvalues[0];
	*(uint4*)&u[2] = *(uint4*)&precalcvalues[2];
	*(uint4*)&u[4] = *(uint4*)&precalcvalues[4];
	*(uint4*)&u[6] = *(uint4*)&precalcvalues[6];
	*(uint4*)&u[8] = *(uint4*)&precalcvalues[8];
	*(uint4*)&u[10] = *(uint4*)&precalcvalues[10];
	*(uint4*)&u[12] = *(uint4*)&precalcvalues[12];
	*(uint4*)&u[14] = *(uint4*)&precalcvalues[14];

	// mix 1
	// Gn1y missing parts
	u[0] = u[0] + u[4] + m;
	u[12] = ROR16(u[12] ^ u[0]);
	u[8] = u[8] + u[12];
	u[4] = ROR2(u[4] ^ u[8], 63U);
	//v[0] = v[0] + v[4] + m64;
	//u[12] = ROR16(u[12] ^ u[0]);
	//v[8] = v[8] + v[12];
	//u[4] = ROR2(u[4] ^ u[8], 63U);
#endif

	// mix 1 remains
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 2
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);

	Gn1x(0, 5, 10, 15, m);
	//Gn1x(0, 5, 10, 15, m64);
	Gn3(              1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 3
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn2(0, 5, 10, 15, 1, 6, 11, 12);
	//Gn1y(2, 7, 8, 13, m64);
	Gn1y(2, 7, 8, 13, m);
	Gn1(3, 4, 9, 14);

	// mix 4
	Gn1(0, 4,  8, 12);
	//Gn1y(1, 5,  9, 13, m64);
	Gn1y(1, 5,  9, 13, m);
	Gn2(                            2, 6, 10, 14, 3, 7, 11, 15);
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 5
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn1y(0, 5, 10, 15, m);
	Gn3(              1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 6
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn3(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13);
	Gn1x(3, 4, 9, 14, m);

	// mix 7
	Gn1(0, 4,  8, 12);
	Gn1x(1, 5, 9, 13, m);
	Gn2(                            2, 6, 10, 14, 3, 7, 11, 15);
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 8
	Gn2(0, 4,  8, 12, 1, 5,  9, 13);
	Gn1y(2, 6, 10, 14, m);
	Gn1(3, 7, 11, 15);
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 9
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn2(0, 5, 10, 15, 1, 6, 11, 12);
	Gn1x(2, 7, 8, 13, m);
	Gn1(3, 4,  9, 14);

	// mix 10
	Gn3(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14);
	Gn1x(3, 7, 11, 15, m);
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 11
	Gn1y(0, 4, 8, 12, m);
	Gn3(              1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn4(0, 5, 10, 15, 1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	// mix 12
	Gn4(0, 4,  8, 12, 1, 5,  9, 13, 2, 6, 10, 14, 3, 7, 11, 15);
	Gn1x(0, 5, 10, 15, m);
	Gn3(              1, 6, 11, 12, 2, 7,  8, 13, 3, 4,  9, 14);

	u[0] = xor3(u[0], hash_h[0], u[8]);
	u[3] = xor3(u[3], hash_h[3], u[11]);

	u[1] = xor3(u[1], hash_h[1], u[9]);
	u[2] = xor3(u[2], hash_h[2], u[10]);

	u[4] = xor3(u[4], hash_h[4], u[12]);
	u[5] = xor3(u[5], hash_h[5], u[13]);
	u[6].x = xor3(u[6].x, hash_h[6].x, u[14].x);

	u32 bucketid;
	u32 bexor = __byte_perm(u[0].x, 0, 0x4012); // first 20 bits
	asm("bfe.u32 %0, %1, 12, 12;" : "=r"(bucketid) : "r"(bexor));
	u32 slotp = atomicAdd(&eq->edata.nslots0[bucketid], 1);
	if (slotp < RB8_NSLOTS)
	{
		slot* __restrict__ s = &eq->round0trees[bucketid][slotp];

		uint4 t1, t2;
		t1.x = __byte_perm(u[0].x, u[0].y, 0x1234);
		t1.y = __byte_perm(u[0].y, u[1].x, 0x1234);
		t1.z = __byte_perm(u[1].x, u[1].y, 0x1234);
		t1.w = __byte_perm(u[1].y, u[2].x, 0x1234);
		*(uint4*)(&s->hash[0]) = t1;

		t2.x = __byte_perm(u[2].x, u[2].y, 0x1234);
		t2.y = __byte_perm(u[2].y, u[3].x, 0x1234);
		t2.z = 0;
		t2.w = block << 1;
		*(uint4*)(&s->hash[4]) = t2;
	}

	bexor = __byte_perm(u[3].x, 0, 0x0123);
	asm("bfe.u32 %0, %1, 12, 12;" : "=r"(bucketid) : "r"(bexor));
	slotp = atomicAdd(&eq->edata.nslots0[bucketid], 1);
	if (slotp < RB8_NSLOTS)
	{
		slot* __restrict__ s = &eq->round0trees[bucketid][slotp];

		uint4 t1, t2;
		t1.x = __byte_perm(u[3].x, u[3].y, 0x2345);
		t1.y = __byte_perm(u[3].y, u[4].x, 0x2345);
		t1.z = __byte_perm(u[4].x, u[4].y, 0x2345);
		t1.w = __byte_perm(u[4].y, u[5].x, 0x2345);
		*(uint4*)(&s->hash[0]) = t1;

		t2.x = __byte_perm(u[5].x, u[5].y, 0x2345);
		t2.y = __byte_perm(u[5].y, u[6].x, 0x2345);
		t2.z = 0;
		t2.w = (block << 1) + 1;
		*(uint4*)(&s->hash[4]) = t2;
	}
}

/*
  Functions digit_1 to digit_8 works by the same principle;
  Each thread does 2-3 slot loads (loads are coalesced). 
  Xorwork of slots is loaded into shared memory and is kept in registers (except for digit_1).
  At the same time, restbits (8 or 9 bits) in xorwork are used for collisions. 
  Restbits determine position in ht.
  Following next is pair creation. First one (or two) pairs' xorworks are put into global memory
  as soon as possible, the rest pairs are saved in shared memory (one u32 per pair - 16 bit indices). 
  In most cases, all threads have one (or two) pairs so with this trick, we offload memory writes a bit in last step.
  In last step we save xorwork of pairs in memory.
*/
template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_1(equi<RB, SM>* eq)
{
	__shared__ u16 ht[256][SSM - 1];
	__shared__ uint2 lastword1[RB8_NSLOTS];
	__shared__ uint4 lastword2[RB8_NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < 256)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;

	const u32 bsize = umin(eq->edata.nslots0[bucketid], RB8_NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	uint2 ta[2];
	uint4 tb[2];

	u32 si[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		const slot* pslot1 = eq->round0trees[bucketid] + si[i];

		// get xhash
		const uint4 a1 = *(uint4*)(&pslot1->hash[0]);
		const uint2 a2 = *(uint2*)(&pslot1->hash[4]);
		ta[i].x = a1.x;
		ta[i].y = a1.y;
		lastword1[si[i]] = ta[i];
		tb[i].x = a1.z;
		tb[i].y = a1.w;
		tb[i].z = a2.x;
		tb[i].w = a2.y;
		lastword2[si[i]] = tb[i];

		asm("bfe.u32 %0, %1, 20, 8;" : "=r"(hr[i]) : "r"(ta[i].x));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	int* pairs = ht_len;

	u32 xors[6];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = ta[i] ^ lastword1[p];

			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				*(uint4*)(&xors[2]) = tb[i] ^ lastword2[p];

				slot &xs = eq->trees[0][xorbucketid][xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
				uint4 ttx;
				ttx.x = xors[5];
				ttx.y = xors[0];
				ttx.z = packer_default::set_bucketid_and_slots(bucketid, si[i], p, 8, RB8_NSLOTS);
				ttx.w = 0;
				*(uint4*)(&xs.hash[4]) = ttx;
			}

			if (pos[i] > 1)
			{
				u32 len = pos[i] - 1;
				u32 pindex = atomicAdd(&pairs_len, len);
				DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_1() overflow MAXPAIRS %d + %d\n", pindex, len);
				len = min(len, MAXPAIRS - pindex);
				#pragma unroll (SSM - 2)
				for (int k = 0; k < len; k++)
				{
					u32 prev = ht[hr[i]][k + 1];
					pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
					if (k + 1 >= len) break;
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = threadid; s < plen; s+= THREADS)
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(RB), "r"(BUCKBITS));
		xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

		if (xorslot < NSLOTS)
		{
			*(uint4*)(&xors[2]) = lastword2[i] ^ lastword2[k];

			slot &xs = eq->trees[0][xorbucketid][xorslot];
			*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
			uint4 ttx;
			ttx.x = xors[5];
			ttx.y = xors[0];
			ttx.z = packer_default::set_bucketid_and_slots(bucketid, i, k, 8, RB8_NSLOTS);
			ttx.w = 0;
			*(uint4*)(&xs.hash[4]) = ttx;
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_2(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][SSM - 1];
	__shared__ u32 lastword1[NSLOTS];
	__shared__ uint4 lastword2[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;

	slot* buck = eq->trees[0][bucketid];
	const u32 bsize = umin(eq->edata.nslots[1][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 ta[2];
	uint4 tt[2];

	u32 si[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		// get slot
		const slot* pslot1 = buck + si[i];

		const uint4 ttx = *(uint4*)(&pslot1->hash[0]);
		lastword1[si[i]] = ta[i] = ttx.x;
		const uint2 tty = *(uint2*)(&pslot1->hash[4]);
		tt[i].x = ttx.y;
		tt[i].y = ttx.z;
		tt[i].z = ttx.w;
		tt[i].w = tty.x;
		lastword2[si[i]] = tt[i];

		hr[i] = tty.y & RESTMASK;
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[5];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			xors[0] = ta[i] ^ lastword1[p];

			xorbucketid = xors[0] >> (12 + RB);
			xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				*(uint4*)(&xors[1]) = tt[i] ^ lastword2[p];
				slotsmall &xs = eq->round2trees[xorbucketid].treessmall[xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
				slottiny &xst = eq->round2trees[xorbucketid].treestiny[xorslot];
				uint2 ttx;
				ttx.x = xors[4];
				ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
				*(uint2*)(&xst.hash[0]) = ttx;
			}

			if (pos[i] > 1)
			{
				u32 len = pos[i] - 1;
				u32 pindex = atomicAdd(&pairs_len, len);
				DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_2() overflow MAXPAIRS %d + %d\n", pindex, len);
				len = min(len, MAXPAIRS - pindex);
				#pragma unroll (SSM - 2)
				for (int k = 0; k < len; k++)
				{
					u32 prev = ht[hr[i]][k + 1];
					pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
					if (k + 1 >= len) break;
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = threadid; s < plen; s+= THREADS)
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		xors[0] = lastword1[i] ^ lastword1[k];

		xorbucketid = xors[0] >> (12 + RB);
		xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
		if (xorslot < NSLOTS)
		{
			*(uint4*)(&xors[1]) = lastword2[i] ^ lastword2[k];
			slotsmall &xs = eq->round2trees[xorbucketid].treessmall[xorslot];
			*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
			slottiny &xst = eq->round2trees[xorbucketid].treestiny[xorslot];
			uint2 ttx;
			ttx.x = xors[4];
			ttx.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
			*(uint2*)(&xst.hash[0]) = ttx;
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_3(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword1[NSLOTS];
	__shared__ u32 lastword2[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;

	const u32 bsize = umin(eq->edata.nslots[2][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];
	u32 ta[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round2trees[bucketid].treessmall[si[i]];
		slottiny &xst = eq->round2trees[bucketid].treestiny[si[i]];

		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword1[si[i]] = tt[i];
		ta[i] = xst.hash[0];
		lastword2[si[i]] = ta[i];
		asm("bfe.u32 %0, %1, 12, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[5];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			xors[4] = ta[i] ^ lastword2[p];

			if (xors[4] != 0)
			{
				*(uint4*)(&xors[0]) = tt[i] ^ lastword1[p];

				bexor = __byte_perm(xors[0], xors[1], 0x2107);
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);

				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->round3trees[xorbucketid].treessmall[xorslot];
					*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
					slottiny &xst = eq->round3trees[xorbucketid].treestiny[xorslot];
					uint2 ttx;
					ttx.x = bexor;
					ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint2*)(&xst.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				u32 len = pos[i] - 1;
				u32 pindex = atomicAdd(&pairs_len, len);
				DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_3() overflow MAXPAIRS %d + %d\n", pindex, len);
				len = min(len, MAXPAIRS - pindex);
				#pragma unroll (SSM -2)
				for (int k = 0; k < len; k++)
				{
					u32 prev = ht[hr[i]][k + 1];
					pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
					if (k + 1 >= len) break;
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = threadid; s < plen; s+= THREADS)
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		xors[4] = lastword2[i] ^ lastword2[k];

		if (xors[4] != 0)
		{
			*(uint4*)(&xors[0]) = lastword1[i] ^ lastword1[k];

			bexor = __byte_perm(xors[0], xors[1], 0x2107);
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->round3trees[xorbucketid].treessmall[xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
				slottiny &xst = eq->round3trees[xorbucketid].treestiny[xorslot];
				uint2 ttx;
				ttx.x = bexor;
				ttx.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
				*(uint2*)(&xst.hash[0]) = ttx;
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_4(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;

	const u32 bsize = umin(eq->edata.nslots[3][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round3trees[bucketid].treessmall[si[i]];
		slottiny &xst = eq->round3trees[bucketid].treestiny[si[i]];

		// get xhash
		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword[si[i]] = tt[i];
		hr[i] = xst.hash[0] & RESTMASK;
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	u32 xors[4];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];

			if (xors[3] != 0)
			{
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(4 + RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[3][xorbucketid][xorslot];
					*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);

					eq->round4bidandsids[xorbucketid][xorslot] = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
				}
			}

			if (pos[i] > 1)
			{
				u32 len = pos[i] - 1;
				u32 pindex = atomicAdd(&pairs_len, len);
				DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_4() overflow MAXPAIRS %d + %d\n", pindex, len);
				len = min(len, MAXPAIRS - pindex);
				#pragma unroll (SSM - 2)
				for (int k = 0; k < len; k++)
				{
					u32 prev = ht[hr[i]][k + 1];
					pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
					if (k + 1 >= len) break;
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = threadid; s < plen; s+= THREADS)
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];
		if (xors[3] != 0)
		{
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(4 + RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->treessmall[3][xorbucketid][xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
				eq->round4bidandsids[xorbucketid][xorslot] = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_5(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;

	slotsmall* buck = eq->treessmall[3][bucketid];
	const u32 bsize = umin(eq->edata.nslots[4][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		tt[i] = *(uint4*)(&pslot1->hash[0]);
		lastword[si[i]] = tt[i];
		asm("bfe.u32 %0, %1, 4, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	u32 xors[4];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];

			if (xors[3] != 0)
			{
				bexor = __byte_perm(xors[0], xors[1], 0x1076);
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[2][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[1];
					ttx.y = xors[2];
					ttx.z = xors[3];
					ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				u32 len = pos[i] - 1;
				u32 pindex = atomicAdd(&pairs_len, len);
				DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_5() overflow MAXPAIRS %d + %d\n", pindex, len);
				len = min(len, MAXPAIRS - pindex);
				#pragma unroll (SSM - 2)
				for (int k = 0; k < len; k++)
				{
					u32 prev = ht[hr[i]][k + 1];
					pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
					if (k + 1 >= len) break;
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = threadid; s < plen; s+= THREADS)
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];

		if (xors[3] != 0)
		{
			bexor = __byte_perm(xors[0], xors[1], 0x1076);
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->treessmall[2][xorbucketid][xorslot];
				uint4 tt;
				tt.x = xors[1];
				tt.y = xors[2];
				tt.z = xors[3];
				tt.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
				*(uint4*)(&xs.hash[0]) = tt;
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_6(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint2 lastword1[NSLOTS];
	__shared__ u32 lastword2[NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
		pairs_len = 0;

	slotsmall* buck = eq->treessmall[2][bucketid];
	const u32 bsize = umin(eq->edata.nslots[5][bucketid], NSLOTS);

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint4 tt[3];

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		tt[i] = *(uint4*)(&pslot1->hash[0]);
		lastword1[si[i]] = *(uint2*)(&tt[i].x);
		lastword2[si[i]] = tt[i].z;
		asm("bfe.u32 %0, %1, 16, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	// doing this to save shared memory
	int* pairs = ht_len;
	__syncthreads();

	u32 xors[3];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			xors[2] = tt[i].z ^ lastword2[p];

			if (xors[2] != 0)
			{
				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

				bexor = __byte_perm(xors[0], xors[1], 0x1076);
				xorbucketid = bexor >> (12 + RB);
				xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[1];
					ttx.y = xors[2];
					ttx.z = bexor;
					ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				xors[2] = tt[i].z ^ lastword2[p];

				if (xors[2] != 0)
				{
					*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

					bexor = __byte_perm(xors[0], xors[1], 0x1076);
					xorbucketid = bexor >> (12 + RB);
					xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
					if (xorslot < NSLOTS)
					{
						slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
						uint4 ttx;
						ttx.x = xors[1];
						ttx.y = xors[2];
						ttx.z = bexor;
						ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						*(uint4*)(&xs.hash[0]) = ttx;
					}
				}

				if (pos[i] > 2)
				{
					u32 len = pos[i] - 2;
					u32 pindex = atomicAdd(&pairs_len, len);
					DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_6() overflow MAXPAIRS %d + %d\n", pindex, len);
					len = min(len, MAXPAIRS - pindex);
					#pragma unroll (SSM - 3)
					for (int k = 0; k < len; k++)
					{
						u32 prev = ht[hr[i]][k + 2];
						pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
						if (k + 1 >= len) break;
					}
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = threadid; s < plen; s+= blockDim.x)
	{
		u32 pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		xors[2] = lastword2[i] ^ lastword2[k];
		if (xors[2] == 0)
			continue;

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

		bexor = __byte_perm(xors[0], xors[1], 0x1076);
		xorbucketid = bexor >> (12 + RB);
		xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
		if (xorslot >= NSLOTS) continue;
		slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
		uint4 ttx;
		ttx.x = xors[1];
		ttx.y = xors[2];
		ttx.z = bexor;
		ttx.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		*(uint4*)(&xs.hash[0]) = ttx;
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_7(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ u32 lastword[NSLOTS][2];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
		pairs_len = 0;

	slotsmall* buck = eq->treessmall[0][bucketid];
	const u32 bsize = umin(eq->edata.nslots[6][bucketid], NSLOTS);

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint4 tt[3];

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		// get xhash
		tt[i] = *(uint4*)(&pslot1->hash[0]);
		*(uint2*)(&lastword[si[i]][0]) = *(uint2*)(&tt[i].x);
		asm("bfe.u32 %0, %1, 12, %2;" : "=r"(hr[i]) : "r"(tt[i].z), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[2];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

			if (xors[1] != 0)
			{
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[0];
					ttx.y = xors[1];
					ttx.z = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					ttx.w = 0;
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

				if (xors[1] != 0)
				{
					asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
					xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
					if (xorslot < NSLOTS)
					{
						slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
						uint4 ttx;
						ttx.x = xors[0];
						ttx.y = xors[1];
						ttx.z = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						ttx.w = 0;
						*(uint4*)(&xs.hash[0]) = ttx;
					}
				}

				if (pos[i] > 2)
				{
					u32 len = pos[i] - 2;
					u32 pindex = atomicAdd(&pairs_len, len);
					DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_7() overflow MAXPAIRS %d + %d\n", pindex, len);
					len = min(len, MAXPAIRS - pindex);
					#pragma unroll (SSM - 3)
					for (int k = 0; k < len; k++)
					{
						u32 prev = ht[hr[i]][k + 2];
						pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
						if (k + 1 >= len) break;
					}
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = threadid; s < plen; s+= blockDim.x)
	{
		int pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = *(uint2*)(&lastword[i][0]) ^ *(uint2*)(&lastword[k][0]);

		if (xors[1] == 0)
			continue;

		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
		xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
		if (xorslot >= NSLOTS) continue;
		slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
		uint4 tt;
		tt.x = xors[0];
		tt.y = xors[1];
		tt.z = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		tt.w = 0;
		*(uint4*)(&xs.hash[0]) = tt;
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_8(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ u32 lastword[NSLOTS][2];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
		pairs_len = 0;

	slotsmall* buck = eq->treessmall[1][bucketid];
	const u32 bsize = umin(eq->edata.nslots[7][bucketid], NSLOTS);

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint2 tt[3];

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		// get xhash
		tt[i] = *(uint2*)(&pslot1->hash[0]);
		*(uint2*)(&lastword[si[i]][0]) = *(uint2*)(&tt[i].x);
		asm("bfe.u32 %0, %1, 8, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[2];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u32 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

			if (xors[1] != 0)
			{
				bexor = __byte_perm(xors[0], xors[1], 0x0765);
				xorbucketid = bexor >> (12 + 8);
				xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
				if (xorslot < RB8_NSLOTS_LD)
				{
					slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
					uint2 tt;
					tt.x = xors[1];
					tt.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint2*)(&xs.hash[0]) = tt;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

				if (xors[1] != 0)
				{
					bexor = __byte_perm(xors[0], xors[1], 0x0765);
					xorbucketid = bexor >> (12 + 8);
					xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
					if (xorslot < RB8_NSLOTS_LD)
					{
						slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
						uint2 tt;
						tt.x = xors[1];
						tt.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						*(uint2*)(&xs.hash[0]) = tt;
					}
				}

				if (pos[i] > 2)
				{
					u32 len = pos[i] - 2;
					u32 pindex = atomicAdd(&pairs_len, len);
					DEBUG_PRINT_IF(pindex + len >= MAXPAIRS, "digit_8() overflow MAXPAIRS %d + %d\n", pindex, len);
					len = min(len, MAXPAIRS - pindex);
					#pragma unroll (SSM - 3)
					for (int k = 0; k < len; k++)
					{
						u32 prev = ht[hr[i]][k + 2];
						pairs[pindex + k] = __byte_perm(si[i], prev, 0x1054);
						if (k + 1 >= len) break;
					}
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	const u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = threadid; s < plen; s+= blockDim.x)
	{
		int pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = *(uint2*)(&lastword[i][0]) ^ *(uint2*)(&lastword[k][0]);

		if (xors[1] == 0)
			continue;

		bexor = __byte_perm(xors[0], xors[1], 0x0765);
		xorbucketid = bexor >> (12 + 8);
		xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
		if (xorslot >= RB8_NSLOTS_LD) continue;
		slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
		uint2 tt;
		tt.x = xors[1];
		tt.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		*(uint2*)(&xs.hash[0]) = tt;
	}
}

/*
  Last round function is similar to previous ones but has different ending.
  We use warps to process final candidates. Each warp process one candidate.
  First two bidandsids (u32 of stored bucketid and two slotids) are retreived by
  lane 0 and lane 16, next four bidandsids by lane 0, 8, 16 and 24, ... until
  all lanes in warp have bidandsids from round 4. Next, each thread retreives
  16 indices. While doing so, indices are put into comparison using atomicExch
  to determine if there are duplicates (tromp's method). At the end, if no
  duplicates are found, candidate solution is saved (all indices). Note that this
  dup check method is not exact so CPU dup checking is needed after.
*/
template <u32 RB, u32 SM, int SSM, u32 FCT, typename PACKER, u32 MAXPAIRS, u32 DUPBITS, u32 W>
__global__ void digit_last_wdc(equi<RB, SM>* eq, u32 nonce)
{
	__shared__ u8 shared_data[8192];
	int* ht_len = (int*)(&shared_data[0]);
	int* pairs = ht_len;
	u32* lastword = (u32*)(&shared_data[256 * 4]);
	u16* ht = (u16*)(&shared_data[256 * 4 + RB8_NSLOTS_LD * 4]);
	u32* pairs_len = (u32*)(&shared_data[8188]);

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
#pragma unroll
	for (u32 i = 0; i < FCT; ++i)
		ht_len[(i * (256 / FCT)) + threadid] = 0;

	if (threadid == ((256 / FCT) - 1))
		*pairs_len = 0;

	slottiny* buck = eq->treestiny[0][bucketid];
	const u32 bsize = umin(eq->edata.nslots8[bucketid], RB8_NSLOTS_LD);

	u32 si[3 * FCT];
	u32 hr[3 * FCT];
	int pos[3 * FCT];
	u32 lw[3 * FCT];
#pragma unroll
	for (u32 i = 0; i < (3 * FCT); ++i)
		pos[i] = SSM;

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < (3 * FCT); ++i)
	{
		si[i] = i * (256 / FCT) + threadid;
		if (si[i] >= bsize) break;

		const slottiny* pslot1 = buck + si[i];

		// get xhash
		uint2 tt = *(uint2*)(&pslot1->hash[0]);
		lw[i] = tt.x;
		lastword[si[i]] = lw[i];

		u32 a;
		asm("bfe.u32 %0, %1, 20, 8;" : "=r"(a) : "r"(lw[i]));
		hr[i] = a;

		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1))
			ht[hr[i] * (SSM - 1) + pos[i]] = si[i];
	}

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < (3 * FCT); ++i)
	{
		if (pos[i] >= SSM) continue;

		for (int k = 0; k < pos[i]; ++k)
		{
			u32 prev = ht[hr[i] * (SSM - 1) + k];
			if (lw[i] != lastword[prev]) continue;
			u32 pindex = atomicAdd(pairs_len, 1);
			if (pindex >= MAXPAIRS) break;
			pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
		}
	}

	__syncthreads();
	const u32 plen = umin(*pairs_len, 64);

#define CALC_LEVEL(a, b, c, d) { \
	u32 plvl = levels[b]; \
	u32* bucks = eq->round4bidandsids[PACKER::get_bucketid(plvl, RB, SM)]; \
	u32 slot1 = PACKER::get_slot1(plvl, RB, SM); \
	u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM); \
	levels[b] = bucks[slot1]; \
	levels[c] = bucks[slot0]; \
				}

#define CALC_LEVEL_SMALL(a, b, c, d) { \
	u32 plvl = levels[b]; \
	slotsmall* bucks = eq->treessmall[a][PACKER::get_bucketid(plvl, RB, SM)]; \
	u32 slot1 = PACKER::get_slot1(plvl, RB, SM); \
	u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM); \
	levels[b] = bucks[slot1].hash[d]; \
	levels[c] = bucks[slot0].hash[d]; \
				}

	u32 lane = threadIdx.x & 0x1f;
	u32 par = threadIdx.x >> 5;

	u32* levels = (u32*)&pairs[MAXPAIRS + ((par & 3) << DUPBITS)];
	u32* susp = levels;

	while (par < plen)
	{
		int pair = pairs[par];
		par += W;

		if (lane % 16 == 0)
		{
			u32 plvl;
			if (lane == 0) plvl = buck[__byte_perm(pair, 0, 0x4510)].hash[1];
			else plvl = buck[__byte_perm(pair, 0, 0x4532)].hash[1];
			slotsmall* bucks = eq->treessmall[1][PACKER::get_bucketid(plvl, RB, SM)];
			u32 slot1 = PACKER::get_slot1(plvl, RB, SM);
			u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM);
			levels[lane] = bucks[slot1].hash[2];
			levels[lane + 8] = bucks[slot0].hash[2];
		}

		if (lane % 8 == 0)
			CALC_LEVEL_SMALL(0, lane, lane + 4, 3);

		if (lane % 4 == 0)
			CALC_LEVEL_SMALL(2, lane, lane + 2, 3);

		if (lane % 2 == 0)
			CALC_LEVEL(0, lane, lane + 1, 4);

		u32 ind[16];

		u32 f1 = levels[lane];
		const slottiny* buck_v4 = &eq->round3trees[PACKER::get_bucketid(f1, RB, SM)].treestiny[0];
		const u32 slot1_v4 = PACKER::get_slot1(f1, RB, SM);
		const u32 slot0_v4 = PACKER::get_slot0(f1, slot1_v4, RB, SM);

		susp[lane] = 0xffffffff;
		susp[32 + lane] = 0xffffffff;

#define CHECK_DUP(a) \
	ANY(atomicExch(&susp[(ind[a] & ((1 << DUPBITS) - 1))], (ind[a] >> DUPBITS)) == (ind[a] >> DUPBITS))

		u32 f2 = buck_v4[slot1_v4].hash[1];
		const slottiny* buck_v3_1 = &eq->round2trees[PACKER::get_bucketid(f2, RB, SM)].treestiny[0];
		const u32 slot1_v3_1 = PACKER::get_slot1(f2, RB, SM);
		const u32 slot0_v3_1 = PACKER::get_slot0(f2, slot1_v3_1, RB, SM);

		susp[64 + lane] = 0xffffffff;
		susp[96 + lane] = 0xffffffff;

		u32 f0 = buck_v3_1[slot1_v3_1].hash[1];
		const slot* buck_v2_1 = eq->trees[0][PACKER::get_bucketid(f0, RB, SM)];
		const u32 slot1_v2_1 = PACKER::get_slot1(f0, RB, SM);
		const u32 slot0_v2_1 = PACKER::get_slot0(f0, slot1_v2_1, RB, SM);

		susp[128 + lane] = 0xffffffff;
		susp[160 + lane] = 0xffffffff;

		u32 f3 = buck_v2_1[slot1_v2_1].hash[6];
		const slot* buck_fin_1 = eq->round0trees[packer_default::get_bucketid(f3, 8, RB8_NSLOTS)];
		const u32 slot1_fin_1 = packer_default::get_slot1(f3, 8, RB8_NSLOTS);
		const u32 slot0_fin_1 = packer_default::get_slot0(f3, slot1_fin_1, 8, RB8_NSLOTS);

		susp[192 + lane] = 0xffffffff;
		susp[224 + lane] = 0xffffffff;

		ind[0] = buck_fin_1[slot1_fin_1].hash[7];
		if (CHECK_DUP(0)) continue;
		ind[1] = buck_fin_1[slot0_fin_1].hash[7];
		if (CHECK_DUP(1)) continue;

		u32 f4 = buck_v2_1[slot0_v2_1].hash[6];
		const slot* buck_fin_2 = eq->round0trees[packer_default::get_bucketid(f4, 8, RB8_NSLOTS)];
		const u32 slot1_fin_2 = packer_default::get_slot1(f4, 8, RB8_NSLOTS);
		const u32 slot0_fin_2 = packer_default::get_slot0(f4, slot1_fin_2, 8, RB8_NSLOTS);

		ind[2] = buck_fin_2[slot1_fin_2].hash[7];
		if (CHECK_DUP(2)) continue;
		ind[3] = buck_fin_2[slot0_fin_2].hash[7];
		if (CHECK_DUP(3)) continue;

		u32 f5 = buck_v3_1[slot0_v3_1].hash[1];
		const slot* buck_v2_2 = eq->trees[0][PACKER::get_bucketid(f5, RB, SM)];
		const u32 slot1_v2_2 = PACKER::get_slot1(f5, RB, SM);
		const u32 slot0_v2_2 = PACKER::get_slot0(f5, slot1_v2_2, RB, SM);

		u32 f6 = buck_v2_2[slot1_v2_2].hash[6];
		const slot* buck_fin_3 = eq->round0trees[packer_default::get_bucketid(f6, 8, RB8_NSLOTS)];
		const u32 slot1_fin_3 = packer_default::get_slot1(f6, 8, RB8_NSLOTS);
		const u32 slot0_fin_3 = packer_default::get_slot0(f6, slot1_fin_3, 8, RB8_NSLOTS);

		ind[4] = buck_fin_3[slot1_fin_3].hash[7];
		if (CHECK_DUP(4)) continue;
		ind[5] = buck_fin_3[slot0_fin_3].hash[7];
		if (CHECK_DUP(5)) continue;

		u32 f7 = buck_v2_2[slot0_v2_2].hash[6];
		const slot* buck_fin_4 = eq->round0trees[packer_default::get_bucketid(f7, 8, RB8_NSLOTS)];
		const u32 slot1_fin_4 = packer_default::get_slot1(f7, 8, RB8_NSLOTS);
		const u32 slot0_fin_4 = packer_default::get_slot0(f7, slot1_fin_4, 8, RB8_NSLOTS);

		ind[6] = buck_fin_4[slot1_fin_4].hash[7];
		if (CHECK_DUP(6)) continue;
		ind[7] = buck_fin_4[slot0_fin_4].hash[7];
		if (CHECK_DUP(7)) continue;

		u32 f8 = buck_v4[slot0_v4].hash[1];
		const slottiny* buck_v3_2 = &eq->round2trees[PACKER::get_bucketid(f8, RB, SM)].treestiny[0];
		const u32 slot1_v3_2 = PACKER::get_slot1(f8, RB, SM);
		const u32 slot0_v3_2 = PACKER::get_slot0(f8, slot1_v3_2, RB, SM);

		u32 f9 = buck_v3_2[slot1_v3_2].hash[1];
		const slot* buck_v2_3 = eq->trees[0][PACKER::get_bucketid(f9, RB, SM)];
		const u32 slot1_v2_3 = PACKER::get_slot1(f9, RB, SM);
		const u32 slot0_v2_3 = PACKER::get_slot0(f9, slot1_v2_3, RB, SM);

		u32 f10 = buck_v2_3[slot1_v2_3].hash[6];
		const slot* buck_fin_5 = eq->round0trees[packer_default::get_bucketid(f10, 8, RB8_NSLOTS)];
		const u32 slot1_fin_5 = packer_default::get_slot1(f10, 8, RB8_NSLOTS);
		const u32 slot0_fin_5 = packer_default::get_slot0(f10, slot1_fin_5, 8, RB8_NSLOTS);

		ind[8] = buck_fin_5[slot1_fin_5].hash[7];
		if (CHECK_DUP(8)) continue;
		ind[9] = buck_fin_5[slot0_fin_5].hash[7];
		if (CHECK_DUP(9)) continue;

		u32 f11 = buck_v2_3[slot0_v2_3].hash[6];
		const slot* buck_fin_6 = eq->round0trees[packer_default::get_bucketid(f11, 8, RB8_NSLOTS)];
		const u32 slot1_fin_6 = packer_default::get_slot1(f11, 8, RB8_NSLOTS);
		const u32 slot0_fin_6 = packer_default::get_slot0(f11, slot1_fin_6, 8, RB8_NSLOTS);

		ind[10] = buck_fin_6[slot1_fin_6].hash[7];
		if (CHECK_DUP(10)) continue;
		ind[11] = buck_fin_6[slot0_fin_6].hash[7];
		if (CHECK_DUP(11)) continue;

		u32 f12 = buck_v3_2[slot0_v3_2].hash[1];
		const slot* buck_v2_4 = eq->trees[0][PACKER::get_bucketid(f12, RB, SM)];
		const u32 slot1_v2_4 = PACKER::get_slot1(f12, RB, SM);
		const u32 slot0_v2_4 = PACKER::get_slot0(f12, slot1_v2_4, RB, SM);

		u32 f13 = buck_v2_4[slot1_v2_4].hash[6];
		const slot* buck_fin_7 = eq->round0trees[packer_default::get_bucketid(f13, 8, RB8_NSLOTS)];
		const u32 slot1_fin_7 = packer_default::get_slot1(f13, 8, RB8_NSLOTS);
		const u32 slot0_fin_7 = packer_default::get_slot0(f13, slot1_fin_7, 8, RB8_NSLOTS);

		ind[12] = buck_fin_7[slot1_fin_7].hash[7];
		if (CHECK_DUP(12)) continue;
		ind[13] = buck_fin_7[slot0_fin_7].hash[7];
		if (CHECK_DUP(13)) continue;

		u32 f14 = buck_v2_4[slot0_v2_4].hash[6];
		const slot* buck_fin_8 = eq->round0trees[packer_default::get_bucketid(f14, 8, RB8_NSLOTS)];
		const u32 slot1_fin_8 = packer_default::get_slot1(f14, 8, RB8_NSLOTS);
		const u32 slot0_fin_8 = packer_default::get_slot0(f14, slot1_fin_8, 8, RB8_NSLOTS);

		ind[14] = buck_fin_8[slot1_fin_8].hash[7];
		if (CHECK_DUP(14)) continue;
		ind[15] = buck_fin_8[slot0_fin_8].hash[7];
		if (CHECK_DUP(15)) continue;

		u32 soli;
		if (lane == 0)
		{
			soli = atomicAdd(&eq->srealcont.nsols, 1);
		}
		soli = SHFL(soli, 0);

		if (soli < MAXREALSOLS)
		{
			u32 pos = lane << 4;
			*(uint4*)(&eq->srealcont.sols[soli][pos]) = *(uint4*)(&ind[0]);
			*(uint4*)(&eq->srealcont.sols[soli][pos + 4]) = *(uint4*)(&ind[4]);
			*(uint4*)(&eq->srealcont.sols[soli][pos + 8]) = *(uint4*)(&ind[8]);
			*(uint4*)(&eq->srealcont.sols[soli][pos + 12]) = *(uint4*)(&ind[12]);
			if (lane == 0)
				eq->srealcont.nonces[soli] = nonce;
		}
	}
}


std::mutex dev_init;
int dev_init_done[8 * 2] = { 0 };

__host__ static int compu32(const void *pa, const void *pb)
{
	uint32_t a = *(uint32_t *)pa, b = *(uint32_t *)pb;
	return a<b ? -1 : a == b ? 0 : +1;
}


__host__ static bool duped(uint32_t* prf)
{
	uint32_t sortprf[512];
	memcpy(sortprf, prf, sizeof(uint32_t) * 512);
	qsort(sortprf, 512, sizeof(uint32_t), &compu32);
	for (uint32_t i = 1; i<512; i++)
		if (sortprf[i] <= sortprf[i - 1])
			return true;
	return false;
}


__host__ __forceinline__ static void sort_pair(uint32_t *a, uint32_t len)
{
	uint32_t    *b = a + len;
	if (a[0] < b[0])
		return;
	for (uint32_t i = 0; i < len; i++)
	{
		uint32_t tmp = a[i];
		a[i] = b[i];
		b[i] = tmp;
	}
}


__host__ static void setheader(blake2b_state *ctx, const char *header, const u32 headerLen, const char* nce, const u32 nonceLen)
{
	uint32_t le_N = WN;
	uint32_t le_K = WK;
	uchar personal[] = "ZcashPoW01230123";
	memcpy(personal + 8, &le_N, 4);
	memcpy(personal + 12, &le_K, 4);
	blake2b_param P[1];
	P->digest_length = HASHOUT;
	P->key_length = 0;
	P->fanout = 1;
	P->depth = 1;
	P->leaf_length = 0;
	P->node_offset = 0;
	P->node_depth = 0;
	P->inner_length = 0;
	memset(P->reserved, 0, sizeof(P->reserved));
	memset(P->salt, 0, sizeof(P->salt));
	memcpy(P->personal, (const uint8_t *)personal, 16);
	blake2b_init_param(ctx, P);
	blake2b_update(ctx, (const uchar *)header, headerLen);
	blake2b_update(ctx, (const uchar *)nce, nonceLen);
}


#ifdef WIN32
typedef hipError_t(CUDAAPI *dec_cuDeviceGet)(hipDevice_t*, int);
typedef hipError_t(CUDAAPI *dec_cuCtxCreate)(hipCtx_t*, unsigned int, hipDevice_t);
typedef hipError_t(CUDAAPI *dec_cuCtxPushCurrent)(hipCtx_t);
typedef hipError_t(CUDAAPI *dec_cuCtxDestroy)(hipCtx_t);

dec_cuDeviceGet _cuDeviceGet = nullptr;
dec_cuCtxCreate _cuCtxCreate = nullptr;
dec_cuCtxPushCurrent _cuCtxPushCurrent = nullptr;
dec_cuCtxDestroy _cuCtxDestroy = nullptr;
#endif


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::eq_cuda_context(int id)
	: device_id(id)
{
	solutions = nullptr;

	dev_init.lock();
	if (!dev_init_done[device_id])
	{
		// only first thread shall init device
		checkCudaErrors(hipSetDevice(device_id));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
		checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

		pctx = nullptr;
	}
	else
	{
		// create new context
		hipDevice_t dev;

#ifdef WIN32
		if (_cuDeviceGet == nullptr)
		{
			HMODULE hmod = LoadLibraryA("nvcuda.dll");
			if (hmod == NULL)
				throw std::runtime_error("Failed to load nvcuda.dll");
			_cuDeviceGet = (dec_cuDeviceGet)GetProcAddress(hmod, "hipDeviceGet");
			if (_cuDeviceGet == nullptr)
				throw std::runtime_error("Failed to get hipDeviceGet address");
			_cuCtxCreate = (dec_cuCtxCreate)GetProcAddress(hmod, "hipCtxCreate");
			if (_cuCtxCreate == nullptr)
				throw std::runtime_error("Failed to get hipCtxCreate address");
			_cuCtxPushCurrent = (dec_cuCtxPushCurrent)GetProcAddress(hmod, "hipCtxPushCurrent");
			if (_cuCtxPushCurrent == nullptr)
				throw std::runtime_error("Failed to get hipCtxPushCurrent address");
			_cuCtxDestroy = (dec_cuCtxDestroy)GetProcAddress(hmod, "hipCtxDestroy");
			if (_cuCtxDestroy == nullptr)
				throw std::runtime_error("Failed to get hipCtxDestroy address");
		}


		checkCudaDriverErrors(_cuDeviceGet(&dev, device_id));
		checkCudaDriverErrors(_cuCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(_cuCtxPushCurrent(pctx));
#else
		checkCudaDriverErrors(hipDeviceGet(&dev, device_id));
		checkCudaDriverErrors(hipCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(hipCtxPushCurrent(pctx));
#endif
	}
	++dev_init_done[device_id];
	dev_init.unlock();

	if (hipMalloc((void**)&device_eq, sizeof(equi<RB, SM>)) != hipSuccess)
		throw std::runtime_error("CUDA: failed to alloc memory");

	solutions = (scontainerreal*)malloc(sizeof(scontainerreal));
}


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ void eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::solve(const char *tequihash_header,
	unsigned int tequihash_header_len,
	const char* nonce,
	unsigned int nonce_len,
	std::function<bool()> cancelf,
	std::function<void(const std::vector<uint32_t>&, size_t, uint32_t, const unsigned char*)> solutionf,
	std::function<void(void)> hashdonef)
{
	blake2b_state blake_ctx;

	int blocks = NBUCKETS;

	setheader(&blake_ctx, tequihash_header, tequihash_header_len, nonce, nonce_len);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_blake_h), &blake_ctx.h, sizeof(u64) * 8, 0, hipMemcpyHostToDevice));
#ifdef PRECALC
	precalc(blake_ctx.h);
#endif

	// todo: improve
	// djezo solver allows last 4 bytes of nonce to be iterrated
	// this can be used to create internal loop - calc initial blake hash only once, then load 8*8 bytes on device (blake state h)
	// then just iterate nn++
	// less CPU load, 1 hipMemcpy less -> faster
	//u32 nn = *(u32*)&nonce[28];
	u32 nn = 0;

	checkCudaErrors(hipMemset(&device_eq->srealcont, 0, sizeof(device_eq->srealcont)));

	for (; nn < 7; nn++)
	{
		checkCudaErrors(hipMemset(&device_eq->edata, 0, sizeof(device_eq->edata)));

		digit_first<RB, SM, PACKER> <<<NBLOCKS / FD_THREADS, FD_THREADS >>>(device_eq, nn);

		digit_1<RB, SM, SSM, PACKER, 2 * NRESTS, 512> <<<4096, 512 >>>(device_eq);

		digit_2<RB, SM, SSM, PACKER, 2 * NRESTS, THREADS> <<<blocks, THREADS >>>(device_eq);

		digit_3<RB, SM, SSM, PACKER, 2 * NRESTS, THREADS> <<<blocks, THREADS >>>(device_eq);

		if (cancelf()) break;

		digit_4<RB, SM, SSM, PACKER, 2 * NRESTS, THREADS> <<<blocks, THREADS >>>(device_eq);

		digit_5<RB, SM, SSM, PACKER, 2 * NRESTS, THREADS> <<<blocks, THREADS >>>(device_eq);

		digit_6<RB, SM, SSM - 1, PACKER, 2 * NRESTS> <<<blocks, NRESTS >>>(device_eq);

		digit_7<RB, SM, SSM - 1, PACKER, 2 * NRESTS> <<<blocks, NRESTS >>>(device_eq);

		digit_8<RB, SM, SSM - 1, PACKER, 2 * NRESTS> <<<blocks, NRESTS >>>(device_eq);

		digit_last_wdc<RB, SM, SSM - 3, 2, PACKER, 64, 8, 4> << <4096, 256 / 2 >> >(device_eq, nn);
	}

	u32 nsols;
	checkCudaErrors(hipMemcpy(&nsols, &device_eq->srealcont.nsols, sizeof(u32), hipMemcpyDeviceToHost));
	if (nsols > 0)
	{
		checkCudaErrors(hipMemcpy(solutions->sols, &device_eq->srealcont.sols, (nsols > MAXREALSOLS ? MAXREALSOLS : nsols) * (512 * 4), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(solutions->nonces, &device_eq->srealcont.nonces, (nsols > MAXREALSOLS ? MAXREALSOLS : nsols) * 4, hipMemcpyDeviceToHost));
	}

	//printf("nsols: %u\n", solutions->nsols);
	//if (solutions->nsols > 9)
	//	printf("missing sol, total: %u\n", solutions->nsols);

	for (u32 s = 0; (s < nsols) && (s < MAXREALSOLS); s++)
	{
		// remove dups on CPU (dup removal on GPU is not fully exact and can pass on some invalid solutions)
		if (duped(solutions->sols[s])) continue;

		// perform sort of pairs
		for (uint32_t level = 0; level < 9; level++)
			for (uint32_t i = 0; i < (1 << 9); i += (2 << level))
				sort_pair(&solutions->sols[s][i], 1 << level);

		std::vector<uint32_t> index_vector(solutions->sols[s], solutions->sols[s] + PROOFSIZE);

		solutionf(index_vector, DIGITBITS, solutions->nonces[s], nullptr);
	}

	hashdonef();
}


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::~eq_cuda_context()
{
	if (solutions)
		free(solutions);

	if (device_eq)
	{
		hipFree(device_eq);
		device_eq = NULL;
	}

	if (pctx)
	{
		// non primary thread, destroy context
#ifdef WIN32
		checkCudaDriverErrors(_cuCtxDestroy(pctx));
#else
		checkCudaDriverErrors(hipCtxDestroy(pctx));
#endif
	}
	else
	{
		checkCudaErrors(hipDeviceReset());

		dev_init_done[device_id] = 0;
	}
}


#ifdef CONFIG_MODE_1
template class eq_cuda_context<CONFIG_MODE_1>;
#endif

#ifdef CONFIG_MODE_2
template class eq_cuda_context<CONFIG_MODE_2>;
#endif

#ifdef CONFIG_MODE_3
template class eq_cuda_context<CONFIG_MODE_3>;
#endif
